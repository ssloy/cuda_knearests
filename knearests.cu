#include "hip/hip_runtime.h"
// Sylvain Lefebvre 2017-10-04
#include <stdio.h>

#include <hip/hip_runtime.h>
#include <assert.h>

#include <set>
#include <map>
#include <vector>
#include <algorithm>
#include <random>
#include <float.h>

// ------------------------------------------------------------

#define KN_kernel        7
#define KN_global        28 // int * KN_kernel
#define POINTS_PER_BLOCK 64

// ------------------------------------------------------------

__device__ int cellFromPoint(int xdim, int ydim, int zdim, float x, float y, float z)
{
  int   i = (int)floor(x * (float)xdim / 1000.f);
  int   j = (int)floor(y * (float)ydim / 1000.f);
  int   k = (int)floor(z * (float)zdim / 1000.f);
  i = max(0, min(i, xdim - 1));
  j = max(0, min(j, ydim - 1));
  k = max(0, min(k, zdim - 1));
  return i + j*xdim + k*xdim*ydim;
}

__global__ void count(const float *points, int numPoints, int xdim, int ydim, int zdim, int *counters)
{
  int id = blockDim.x * blockIdx.x + threadIdx.x;
  if (id < numPoints) {
    float x = points[id * 3 + 0];
    float y = points[id * 3 + 1];
    float z = points[id * 3 + 2];
    int cell = cellFromPoint(xdim, ydim, zdim, x, y, z);
    atomicAdd(counters + cell, 1);
  }
}

__global__ void reserve(int xdim, int ydim, int zdim, const int *counters, int *globalcounter, int *ptrs)
{
  int id = blockDim.x * blockIdx.x + threadIdx.x;
  if (id < xdim*ydim*zdim) {
    int cnt = counters[id];
    if (cnt > 0) {
      ptrs[id] = 1 + atomicAdd(globalcounter, cnt); // adding 1 so that null tags empty
    }
  }
}

__global__ void store(const float *points, int numPoints, int xdim, int ydim, int zdim, const int *ptrs, int *counters, int num_stored, float *stored_points)
{
  int id = blockDim.x * blockIdx.x + threadIdx.x;
  if (id < numPoints) {
    float x = points[id * 3 + 0];
    float y = points[id * 3 + 1];
    float z = points[id * 3 + 2];
    int cell = cellFromPoint(xdim, ydim, zdim, x, y, z);
    int pos = ptrs[cell] + atomicAdd(counters + cell, 1);
    stored_points[pos * 3 + 0] = x;
    stored_points[pos * 3 + 1] = y;
    stored_points[pos * 3 + 2] = z;
  }
}

// Launch one per page, grouped by page size
__global__ void knearest(
  int xdim, int ydim, int zdim,
  int num_stored, const int *ptrs, const int *counters, const float *stored_points,
  int num_cell_offsets, const int *cell_offsets, const float *cell_offset_distances,
  unsigned int *g_knearests
  )
{
  float previous_max_dist = -1.0f;
  int   previous_max_id = -1;

  int point_in = 1 + threadIdx.x + blockIdx.x * POINTS_PER_BLOCK;

  if (point_in >= num_stored) return;

  // point considered by this thread
  float x = stored_points[point_in * 3 + 0];
  float y = stored_points[point_in * 3 + 1];
  float z = stored_points[point_in * 3 + 2];

  // bool empty_point = (x == 0.0f && y == 0.0f && z == 0.0f); // no longer possible as 'page size == 1'

  int cell_in = cellFromPoint(xdim, ydim, zdim, x, y, z);

  for (int knpass = 0; knpass < KN_global / KN_kernel; knpass++) {

    // each thread updates its k-nearests
#if 1
    int offs = threadIdx.x * KN_kernel;
    __shared__ unsigned int knearests[KN_kernel * POINTS_PER_BLOCK];
    __shared__ float        knearests_dists[KN_kernel * POINTS_PER_BLOCK];
#else
    const int offs = 0;
    unsigned int knearests      [KN_kernel];
    float        knearests_dists[KN_kernel];
#endif
    for (int i = 0; i < KN_kernel; i++) {
      knearests[offs + i] = UINT_MAX;
      knearests_dists[offs + i] = FLT_MAX;
    }
    int   knearests_prev_max_k = 0;
    float knearests_prev_max_d = FLT_MAX;
    int   knearests_prev_max_id = INT_MAX;

    for (int o = 0; o < num_cell_offsets; o++)  {

      float min_dist = cell_offset_distances[o];
      bool kn_completed = (knearests_prev_max_d < min_dist);

      if (kn_completed) break;

      int cell = cell_in + cell_offsets[o];
      if (cell >= 0 && cell < xdim*ydim*zdim) {

        int cell_base = ptrs[cell];

        int num = counters[cell];
        // go through all pages of this cell
        for (int ptr = cell_base; ptr < cell_base + num; ptr++) {

          // read point
          float x_cmp = stored_points[ptr * 3 + 0];
          float y_cmp = stored_points[ptr * 3 + 1];
          float z_cmp = stored_points[ptr * 3 + 2];

          float d = (x_cmp - x)*(x_cmp - x) + (y_cmp - y)*(y_cmp - y) + (z_cmp - z)*(z_cmp - z);

          if ((d < knearests_prev_max_d || (d == knearests_prev_max_d && ptr < knearests_prev_max_id))
            && (d > previous_max_dist || (d == previous_max_dist    && ptr > previous_max_id))) {
            // replace current max
            knearests[offs + knearests_prev_max_k] = ptr;
            knearests_dists[offs + knearests_prev_max_k] = d;
            // find out new max
            knearests_prev_max_d = -1.0f;
            knearests_prev_max_id = -1;
            for (int k = 0; k < KN_kernel; k++) {
              if (knearests_dists[offs + k] > knearests_prev_max_d
                || (knearests_dists[offs + k] == knearests_prev_max_d && knearests[offs + k] > knearests_prev_max_id)) {
                knearests_prev_max_k = k;
                knearests_prev_max_d = knearests_dists[offs + k];
                knearests_prev_max_id = knearests[offs + k];
              }
            }
          }
        }

      } // valid cell id
    } // offsets

    // store result
    for (int i = 0; i < KN_kernel; i++) {
      g_knearests[point_in + (knpass * KN_kernel + i) * num_stored] = knearests[offs + i];
    }
    previous_max_dist = knearests_dists[offs + knearests_prev_max_k];
    previous_max_id = knearests[offs + knearests_prev_max_k];
  }
}

// ------------------------------------------------------------

typedef struct {
  int K;
  int dimx, dimy, dimz;
  int num_cell_offsets;
  int allocated_points;
  int *d_cell_offsets;
  float *d_cell_offset_dists;
  int *d_counters;
  int *d_ptrs;
  int *d_globcounter;
  float *d_stored_points;
  unsigned int *d_knearests;
} kn_problem;

// ------------------------------------------------------------

void kn_firstbuild(kn_problem *kn,float *d_points, int numpoints)
{
  hipError_t err = hipSuccess;
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipEventRecord(start);

  // count points per grid cell
  {
    int threadsPerBlock = 256;
    int blocksPerGrid = (numpoints + threadsPerBlock - 1) / threadsPerBlock;
    count << <blocksPerGrid, threadsPerBlock >> >(d_points, numpoints, kn->dimx, kn->dimy, kn->dimz, kn->d_counters);
    err = hipGetLastError();
    if (err != hipSuccess) {
      fprintf(stderr, "Failed (1) (error code %s)!\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
    }
  }

  // reserve memory for stored points
  {
    int threadsPerBlock = 256;
    int blocksPerGrid = (kn->dimx*kn->dimy*kn->dimz + threadsPerBlock - 1) / threadsPerBlock;
    reserve << <blocksPerGrid, threadsPerBlock >> >(kn->dimx, kn->dimy, kn->dimz, kn->d_counters, kn->d_globcounter, kn->d_ptrs);
    err = hipGetLastError();
    if (err != hipSuccess) {
      fprintf(stderr, "Failed (2) (error code %s)!\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
    }
  }

#if 0
  // count requied point storage
  int globcounter = 0;
  hipMemcpy(&globcounter, kn->d_globcounter, sizeof(int), hipMemcpyDeviceToHost);
  fprintf(stderr, "max storage position: %d (%d points allocated) \n", 1 + globcounter, kn->allocated_points);
#endif

  // store
  {
    // zero counters
    hipMemset(kn->d_counters, 0x00, kn->dimx*kn->dimy*kn->dimz*sizeof(int));
    // call kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (numpoints + threadsPerBlock - 1) / threadsPerBlock;
    store << <blocksPerGrid, threadsPerBlock >> >(d_points, numpoints, kn->dimx, kn->dimy, kn->dimz, kn->d_ptrs, kn->d_counters, kn->allocated_points, kn->d_stored_points);
    err = hipGetLastError();
    if (err != hipSuccess) {
      fprintf(stderr, "Failed (3) (error code %s)!\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
    }
  }

  hipEventRecord(stop);

  hipEventSynchronize(stop);
  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);
  fprintf(stderr, "kn_firstbuild: %f msec\n", milliseconds);
}

// ------------------------------------------------------------

kn_problem *kn_prepare(float *points, int numpoints)
{
  hipError_t err = hipSuccess;

  kn_problem *kn = (kn_problem*)malloc(sizeof(kn_problem));
  kn->K = KN_global;

  int sz = max(1,(int)round(pow(numpoints / 11.3f, 1.0f / 3.0)));
  kn->dimx = sz;
  kn->dimy = sz;
  kn->dimz = sz;

  // create cell offsets
  int Nmax = 8;
  if (sz < Nmax) {
    fprintf(stderr, "The current implementation does not support low number of input points\n");
    exit(EXIT_FAILURE);
  }
  // very naive approach, that's all fine, pre-computed once
  int alloc = Nmax*Nmax*Nmax*Nmax;
  int   *cell_offsets = (int*)malloc(alloc*sizeof(int));
  float *cell_offset_dists = (float*)malloc(alloc*sizeof(float));
  cell_offsets[0] = 0;
  cell_offset_dists[0] = 0.0f;
  kn->num_cell_offsets = 1;
  Nmax = min(sz, Nmax);
  for (int ring = 1; ring < Nmax; ring++) {
    for (int k = -Nmax / 2; k <= Nmax / 2; k++) {
      for (int j = -Nmax / 2; j <= Nmax / 2; j++) {
        for (int i = -Nmax / 2; i <= Nmax / 2; i++) {
          if (max(abs(i), max(abs(j), abs(k))) == ring) {
            int id_offset = i + j*kn->dimx + k*kn->dimx*kn->dimy;
            if (id_offset == 0) { 
              fprintf(stderr, "Error generating offsets\n");
              exit(EXIT_FAILURE); 
            }
            cell_offsets[kn->num_cell_offsets] = id_offset;
            float d = (float)(ring - 1) / (float)max(kn->dimx, max(kn->dimy, kn->dimz));
            cell_offset_dists[kn->num_cell_offsets] = d*d; // squared
            kn->num_cell_offsets++;
            if (kn->num_cell_offsets >= alloc) {
              exit(EXIT_FAILURE);
            }
          }
        }
      }
    }
  }

  // allocate cell offsets
  kn->d_cell_offsets = NULL;
  err = hipMalloc((void **)&kn->d_cell_offsets, kn->num_cell_offsets*sizeof(int));
  if (err != hipSuccess) {
    fprintf(stderr, "Failed (1) to allocate (error code %s)!\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }
  err = hipMemcpy(kn->d_cell_offsets, cell_offsets, kn->num_cell_offsets*sizeof(int), hipMemcpyHostToDevice);
  if (err != hipSuccess) {
    fprintf(stderr, "Failed (2) to copy from host to device (error code %s)!\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }
  free(cell_offsets);

  // allocate cell offset distances
  kn->d_cell_offset_dists = NULL;
  err = hipMalloc((void **)&kn->d_cell_offset_dists, kn->num_cell_offsets*sizeof(float));
  if (err != hipSuccess) {
    fprintf(stderr, "Failed (3) to allocate (error code %s)!\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }
  err = hipMemcpy(kn->d_cell_offset_dists, cell_offset_dists, kn->num_cell_offsets*sizeof(float), hipMemcpyHostToDevice);
  if (err != hipSuccess) {
    fprintf(stderr, "Failed (4) to copy from host to device (error code %s)!\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }
  free(cell_offset_dists);

  // allocate input points
  float *d_points = NULL;
  err = hipMalloc((void **)&d_points, numpoints * 3 * sizeof(float));
  if (err != hipSuccess) {
    fprintf(stderr, "Failed (5) to allocate (error code %s)!\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }
  err = hipMemcpy(d_points, points, numpoints * 3 * sizeof(float), hipMemcpyHostToDevice);
  if (err != hipSuccess) {
    fprintf(stderr, "Failed (6) to copy from host to device (error code %s)!\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }

  // allocate cell counters
  kn->d_counters = NULL;
  err = hipMalloc((void **)&kn->d_counters, kn->dimx*kn->dimy*kn->dimz*sizeof(int));
  if (err != hipSuccess) {
    fprintf(stderr, "Failed (7) to allocate (%s)!\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }
  hipMemset(kn->d_counters, 0x00, kn->dimx*kn->dimy*kn->dimz*sizeof(int));

  // allocate cell start pointers
  kn->d_ptrs = NULL;
  err = hipMalloc((void **)&kn->d_ptrs, kn->dimx*kn->dimy*kn->dimz*sizeof(int));
  if (err != hipSuccess) {
    fprintf(stderr, "Failed (8) to allocate (%s)!\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }
  hipMemset(kn->d_ptrs, 0x00, kn->dimx*kn->dimy*kn->dimz*sizeof(int));

  // allocate global counter for allocation
  kn->d_globcounter = NULL;
  err = hipMalloc((void **)&kn->d_globcounter, sizeof(int));
  if (err != hipSuccess) {
    fprintf(stderr, "Failed (9) to allocate (%s)!\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }
  hipMemset(kn->d_globcounter, 0x00, sizeof(int));

  // allocate stored points
  kn->allocated_points = numpoints + 1;
  kn->d_stored_points = NULL;
  err = hipMalloc((void **)&kn->d_stored_points, kn->allocated_points * sizeof(float) * 3);
  hipMemset(kn->d_stored_points, 0x00, kn->allocated_points * sizeof(float) * 3);
  if (err != hipSuccess) {
    fprintf(stderr, "Failed (10) to allocate (error code %s)!\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }

  // allocate knearests
  kn->d_knearests = NULL;
  err = hipMalloc((void **)&kn->d_knearests, kn->allocated_points * KN_global * sizeof(int));
  hipMemset(kn->d_knearests, 0xFF, kn->allocated_points * KN_global * sizeof(int));
  if (err != hipSuccess) {
    fprintf(stderr, "Failed (11) to allocate (error code %s)!\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }

  // construct initial structure
  kn_firstbuild(kn,d_points,numpoints);

  // we no longer need the initial points
  hipFree(d_points);

  return kn;
}

// ------------------------------------------------------------

void kn_solve(kn_problem *kn)
{
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipError_t err = hipSuccess;

  int threadsPerBlock = POINTS_PER_BLOCK;
  int blocksPerGrid = (kn->allocated_points + threadsPerBlock - 1) / POINTS_PER_BLOCK;

  hipEventRecord(start);

  knearest << <blocksPerGrid, threadsPerBlock >> >(
    kn->dimx, kn->dimy, kn->dimz, kn->allocated_points,
    kn->d_ptrs, kn->d_counters, kn->d_stored_points,
    kn->num_cell_offsets, kn->d_cell_offsets, kn->d_cell_offset_dists,
    kn->d_knearests);
  err = hipGetLastError();
  if (err != hipSuccess) {
    fprintf(stderr, "Failed  (error code %s)!\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }

  hipEventRecord(stop);

  hipEventSynchronize(stop);
  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);
  fprintf(stderr,"kn_solve: %f msec\n", milliseconds);
}

// ------------------------------------------------------------

void kn_free(kn_problem **kn)
{
  hipFree((*kn)->d_cell_offsets);
  hipFree((*kn)->d_cell_offset_dists);
  hipFree((*kn)->d_counters);
  hipFree((*kn)->d_ptrs);
  hipFree((*kn)->d_globcounter);
  hipFree((*kn)->d_stored_points);
  hipFree((*kn)->d_knearests);
  free(*kn);
  *kn = NULL;
}

// ------------------------------------------------------------

float *kn_get_points(kn_problem *kn)
{
  float *stored_points = (float*)malloc(kn->allocated_points * sizeof(float) * 3);
  hipError_t err = hipMemcpy(stored_points, kn->d_stored_points, kn->allocated_points * sizeof(float) * 3, hipMemcpyDeviceToHost);
  if (err != hipSuccess) {
    fprintf(stderr, "[kn_get_points] Failed to copy from device to host (error code %s)!\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }
  return stored_points;
}

// ------------------------------------------------------------

unsigned int *kn_get_knearests(kn_problem *kn)
{
  unsigned int *knearests = (unsigned int*)malloc(kn->allocated_points * KN_global * sizeof(int));
  hipError_t err = hipMemcpy(knearests, kn->d_knearests, kn->allocated_points * KN_global * sizeof(int), hipMemcpyDeviceToHost);
  if (err != hipSuccess) {
    fprintf(stderr, "[kn_get_knearests] Failed to copy from device to host (error code %s)!\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }
  return knearests;
}

// ------------------------------------------------------------

void kn_sanity_check(kn_problem *kn)
{
  hipError_t err = hipSuccess;

  float *stored_points = (float*)malloc(kn->allocated_points * sizeof(float) * 3);
  err = hipMemcpy(stored_points, kn->d_stored_points, kn->allocated_points * sizeof(float) * 3, hipMemcpyDeviceToHost);
  if (err != hipSuccess) {
    fprintf(stderr, "[kn_sanity_check:1] Failed to copy from device to host (error code %s)!\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }

  unsigned int *knearests = (unsigned int*)malloc(kn->allocated_points * KN_global * sizeof(int));
  err = hipMemcpy(knearests, kn->d_knearests, kn->allocated_points * KN_global * sizeof(int), hipMemcpyDeviceToHost);
  if (err != hipSuccess) {
    fprintf(stderr, "[kn_sanity_check:2] Failed to copy from device to host (error code %s)!\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }

  int *counters = (int*)malloc(kn->dimx*kn->dimy*kn->dimz*sizeof(int));
  err = hipMemcpy(counters, kn->d_counters, kn->dimx*kn->dimy*kn->dimz*sizeof(int), hipMemcpyDeviceToHost);
  if (err != hipSuccess) {
    fprintf(stderr, "[kn_sanity_check:3] Failed to copy from device to host (error code %s)!\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }

  int *ptrs = (int*)malloc(kn->dimx*kn->dimy*kn->dimz*sizeof(int));
  err = hipMemcpy(ptrs, kn->d_ptrs, kn->dimx*kn->dimy*kn->dimz*sizeof(int), hipMemcpyDeviceToHost);
  if (err != hipSuccess) {
    fprintf(stderr, "[kn_sanity_check:4] Failed to copy from device to host (error code %s)!\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }

  // stats on counters
  int tot = 0;
  int cmin = INT_MAX, cmax = 0;
  std::map<int, int> histo;
  for (int c = 0; c < kn->dimx*kn->dimy*kn->dimz; c++) {
    histo[counters[c]]++;
    cmin = min(cmin, counters[c]);
    cmax = max(cmax, counters[c]);
    tot += counters[c];
  }
  printf("Grid:  points per cell: %d (min), %d (max), %f avg, total %d\n", cmin, cmax, (kn->allocated_points-1) / (float)(kn->dimx*kn->dimy*kn->dimz), tot);
  for (auto H : histo) {
    fprintf(stderr, "[%d] => %d\n", H.first, H.second);
  }

  std::minstd_rand rnd;
  int r = rnd();
  for (int tests = 0; tests < kn->allocated_points-1; tests++) {
    int allp = 1+((tests + r) % (kn->allocated_points-1));
  // for (int allp = 1; allp < kn->allocated_points; allp++) {
    //for (int allp = kn->allocated_points - 1; allp >= 0; allp--) {
    fprintf(stderr, "%d/%d ", allp, kn->allocated_points);
    float x = stored_points[allp * 3 + 0], y = stored_points[allp * 3 + 1], z = stored_points[allp * 3 + 2];
    //if (x == 0.0f && y == 0.0f && z == 0.0f) continue;
    // sanity check
    std::set<int> kns;
    for (int i = 0; i < KN_global; ++i) {
      int kni = knearests[allp + i*kn->allocated_points];
      if (kni < UINT_MAX) {
        if (kns.find(kni) != kns.end()) {
          for (int i = 0; i < KN_global; i++) {
            int kni = knearests[allp + i*kn->allocated_points];
            if (kni < UINT_MAX) {
              float kx = stored_points[kni * 3 + 0];
              float ky = stored_points[kni * 3 + 1];
              float kz = stored_points[kni * 3 + 2];
              int ci = (int)floor(kx*kn->dimx);
              int cj = (int)floor(ky*kn->dimy);
              int ck = (int)floor(kz*kn->dimz);
              float d = (x - kx)*(x - kx) + (y - ky)*(y - ky) + (z - kz)*(z - kz);
              fprintf(stderr, "   (%d) %d (%f,%f,%f) [%d,%d,%d] \t=> %f\n", i, kni, kx, ky, kz, ci, cj, ck, d);
            }
          }
          fprintf(stderr, "ERROR duplicated entry %d\n", kni);
          exit(EXIT_FAILURE);
        }
        kns.insert(kni);
      }
    }
    // now brute force search
    std::vector<std::pair<float, int> > kn_check;
    for (int c = 1; c < kn->allocated_points; c++) {
      float kx = stored_points[c * 3 + 0];
      float ky = stored_points[c * 3 + 1];
      float kz = stored_points[c * 3 + 2];
      // if (kx == 0.0f && ky == 0.0f && kz == 0.0f) continue;
      float d = (x - kx)*(x - kx) + (y - ky)*(y - ky) + (z - kz)*(z - kz);
      kn_check.push_back(std::make_pair(d, c));
      if (kn_check.size() > 100000) {
        std::sort(kn_check.begin(), kn_check.end());
        kn_check.resize(KN_global);
      }
    }
    std::sort(kn_check.begin(), kn_check.end());
    kn_check.resize(min((int)kn_check.size(),KN_global));
    for (int k = 0; k < (int)kn_check.size(); k++) {
      float kx = stored_points[kn_check[k].second * 3 + 0];
      float ky = stored_points[kn_check[k].second * 3 + 1];
      float kz = stored_points[kn_check[k].second * 3 + 2];
      int ci = (int)floor(kx*kn->dimx);
      int cj = (int)floor(ky*kn->dimy);
      int ck = (int)floor(kz*kn->dimz);
      if (kns.find(kn_check[k].second) == kns.end()) {
        // dump current configuration
        int pi = (int)floor(x*kn->dimx);
        int pj = (int)floor(y*kn->dimy);
        int pk = (int)floor(z*kn->dimz);
        fprintf(stderr, "============== (%f,%f,%f) [%d,%d,%d] (counter:%d ptr:%d)\n",
          x, y, z,
          pi, pj, pk,
          counters[pi + pj*kn->dimx + pk*kn->dimx*kn->dimy],
          ptrs[pi + pj*kn->dimx + pk*kn->dimx*kn->dimy]);
        for (int i = 0; i < KN_global; i++) {
          int kni = knearests[allp + i*kn->allocated_points];
          if (kni < UINT_MAX) {
            float kx = stored_points[kni * 3 + 0];
            float ky = stored_points[kni * 3 + 1];
            float kz = stored_points[kni * 3 + 2];
            int ci = (int)floor(kx*kn->dimx);
            int cj = (int)floor(ky*kn->dimy);
            int ck = (int)floor(kz*kn->dimz);
            float d = (x - kx)*(x - kx) + (y - ky)*(y - ky) + (z - kz)*(z - kz);
            fprintf(stderr, "   (%d) %d (%f,%f,%f) [%d,%d,%d] \t=> %f\n", i, kni, kx, ky, kz, ci, cj, ck, d);
          }
        }
        fprintf(stderr, "ERROR cannot find knearest %d SANITY CHECK FAILED\n", kn_check[k].second);
        float d = (x - kx)*(x - kx) + (y - ky)*(y - ky) + (z - kz)*(z - kz);
        fprintf(stderr, "**** [%d] (%f,%f,%f) [%d,%d,%d]  %d => %f (%f)\n", k, kx, ky, kz, ci, cj, ck, kn_check[k].second, kn_check[k].first, d);
        exit(EXIT_FAILURE);
      }
    }
    fprintf(stderr, " [ok]\n");
  }

  free(knearests);
  free(ptrs);
  free(counters);
  free(stored_points);

}

// ------------------------------------------------------------

int kn_num_points(kn_problem *kn)
{
  return kn->allocated_points - 1;
}


// ------------------------------------------------------------

typedef struct {
  float        *points;
  unsigned int *kns;
  int           K;
  int           allocated_points;
  int           point_id;
  int           k_rank;
} kn_iterator;

// ------------------------------------------------------------

kn_iterator  *kn_begin_enum(kn_problem *kn)
{
  kn_iterator *it = (kn_iterator*)malloc(sizeof(kn_iterator));
  it->K = kn->K;
  it->points = kn_get_points(kn);
  it->kns = kn_get_knearests(kn);
  it->allocated_points = kn->allocated_points;
  it->k_rank = -1;
  it->point_id = -1;
  return it;
}

// ------------------------------------------------------------

float        *kn_point(kn_iterator *it,int point_id)
{
  return it->points + (point_id+1)*3;
}

// ------------------------------------------------------------

float        *kn_first_nearest(kn_iterator *it, int point_id)
{
  it->point_id = point_id + 1;
  it->k_rank = 0;
  unsigned int kid = it->kns[it->point_id + it->k_rank*it->allocated_points];
  if (kid < UINT_MAX) {
    return it->points + kid*3;
  } else {
    return NULL;
  }
}

// ------------------------------------------------------------

float        *kn_next_nearest(kn_iterator *it)
{
  it->k_rank++;
  if (it->k_rank >= it->K) return NULL;
  unsigned int kid = it->kns[it->point_id + it->k_rank*it->allocated_points];
  if (kid < UINT_MAX) {
    return it->points + kid * 3;
  } else {
    return NULL;
  }
}

// ------------------------------------------------------------

void          kn_end_enum(kn_iterator **it)
{
  free((*it)->points);
  free((*it)->kns);
  free(*it);
  *it = NULL;
}

// ------------------------------------------------------------
