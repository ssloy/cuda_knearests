#include "hip/hip_runtime.h"
#include <string>
#include <vector>
#include <iostream>
#include <iomanip>
#include <fstream>
#include <cmath>
#include <hip/hip_runtime.h>

#include "stopwatch.h"
#include "knearests.h"
#include "voronoi.h"

__device__ float4 point_from_ptr3(float* f) {
    return make_float4(f[0], f[1], f[2], 1);
}
__device__ float4 minus4(float4 A, float4 B) {
    return make_float4(A.x-B.x, A.y-B.y, A.z-B.z, A.w-B.w);
}
__device__ float4 plus4(float4 A, float4 B) {
    return make_float4(A.x+B.x, A.y+B.y, A.z+B.z, A.w+B.w);
}
__device__ float dot4(float4 A, float4 B) {
    return A.x*B.x + A.y*B.y + A.z*B.z + A.w*B.w;
}
__device__ float dot3(float4 A, float4 B) {
    return A.x*B.x + A.y*B.y + A.z*B.z;
}
__device__ float4 mul3(float s, float4 A) {
    return make_float4(s*A.x, s*A.y, s*A.z, 1.);
}
__device__ float4 cross3(float4 A, float4 B) {
    return make_float4(A.y*B.z - A.z*B.y, A.z*B.x - A.x*B.z, A.x*B.y - A.y*B.x, 0);
}
__device__ float4 plane_from_point_and_normal(float4 P, float4 n) {
    return  make_float4(n.x, n.y, n.z, -dot3(P, n));
}
__device__ inline float det2x2(float a11, float a12, float a21, float a22) {
    return a11*a22 - a12*a21;
}
__device__ inline float det3x3(float a11, float a12, float a13, float a21, float a22, float a23, float a31, float a32, float a33) {
    return a11*det2x2(a22, a23, a32, a33) - a21*det2x2(a12, a13, a32, a33) + a31*det2x2(a12, a13, a22, a23);
}

__device__ inline float get_tet_volume(float4 A, float4 B, float4 C) {
    return -det3x3(A.x, A.y, A.z, B.x, B.y, B.z, C.x, C.y, C.z)/6.;
}
__device__ void get_tet_volume_and_barycenter(float4& bary, float& volume, float4 A, float4 B, float4 C, float4 D) {
    volume = get_tet_volume(minus4(A, D), minus4(B, D), minus4(C, D));
    bary = make_float4(.25*(A.x+B.x+C.x+D.x), .25*(A.y+B.y+C.y+D.y), .25*(A.z+B.z+C.z+D.z), 1);
}
__device__ float4 project_on_plane(float4 P, float4 plane) {
    float4 n = make_float4(plane.x, plane.y, plane.z, 0);
    float lambda = (dot4(n, P) + plane.w)/dot4(n, n);
    //    lambda = (dot3(n, P) + plane.w) / norm23(n);
    return plus4(P, mul3(-lambda, n));
}
template <typename T> __device__ void inline swap(T& a, T& b) { T c(a); a = b; b = c; }


__device__ ConvexCell::ConvexCell(int p_seed, float* p_pts,Status *p_status) {
    float eps  = .1f;
    float xmin = -eps;
    float ymin = -eps;
    float zmin = -eps;
    float xmax = 1000 + eps;
    float ymax = 1000 + eps;
    float zmax = 1000 + eps;
    pts = p_pts;
    first_boundary_ = END_OF_LIST;
    FOR(i, _MAX_P_) boundary_next(i) = END_OF_LIST;
    voro_id = p_seed;
    voro_seed = make_float4(pts[3 * voro_id], pts[3 * voro_id + 1], pts[3 * voro_id + 2], 1);
    status = p_status;
    *status = success;

    clip(0) = make_float4( 1.0,  0.0,  0.0, -xmin);
    clip(1) = make_float4(-1.0,  0.0,  0.0,  xmax);
    clip(2) = make_float4( 0.0,  1.0,  0.0, -ymin);
    clip(3) = make_float4( 0.0, -1.0,  0.0,  ymax);
    clip(4) = make_float4( 0.0,  0.0,  1.0, -zmin);
    clip(5) = make_float4( 0.0,  0.0, -1.0,  zmax);
    nb_v = 6;

    tr(0) = make_uchar3(2, 5, 0);
    tr(1) = make_uchar3(5, 3, 0);
    tr(2) = make_uchar3(1, 5, 2);
    tr(3) = make_uchar3(5, 1, 3);
    tr(4) = make_uchar3(4, 2, 0);
    tr(5) = make_uchar3(4, 0, 3);
    tr(6) = make_uchar3(2, 4, 1);
    tr(7) = make_uchar3(4, 3, 1);
    nb_t = 8;
}

__device__  bool ConvexCell::is_security_radius_reached(float4 last_neig) {
    // finds furthest voro vertex distance2
    float v_dist = 0;
    FOR(i, nb_t) {
        float4 pc = compute_triangle_point(tr(i));
        float4 diff = minus4(pc, voro_seed);
        float d2 = dot3(diff, diff); // TODO safe to put dot4 here, diff.w = 0
        v_dist = max(d2, v_dist);
    }
    //compare to new neighbors distance2
    float4 diff = minus4(last_neig, voro_seed); // TODO it really should take index of the neighbor instead of the float4, then would be safe to put dot4
    float d2 = dot3(diff, diff);
    return (d2 > 4*v_dist);
}

__device__ inline  uchar& ConvexCell::ith_plane(uchar t, int i) {
    return reinterpret_cast<uchar *>(&(tr(t)))[i];
}

__device__ float4 ConvexCell::compute_triangle_point(uchar3 t, bool persp_divide) const {
    float4 pi1 = clip(t.x);
    float4 pi2 = clip(t.y);
    float4 pi3 = clip(t.z);
    float4 result;
    result.x = -det3x3(pi1.w, pi1.y, pi1.z, pi2.w, pi2.y, pi2.z, pi3.w, pi3.y, pi3.z);
    result.y = -det3x3(pi1.x, pi1.w, pi1.z, pi2.x, pi2.w, pi2.z, pi3.x, pi3.w, pi3.z);
    result.z = -det3x3(pi1.x, pi1.y, pi1.w, pi2.x, pi2.y, pi2.w, pi3.x, pi3.y, pi3.w);
    result.w =  det3x3(pi1.x, pi1.y, pi1.z, pi2.x, pi2.y, pi2.z, pi3.x, pi3.y, pi3.z);
    if (persp_divide) return make_float4(result.x / result.w, result.y / result.w, result.z / result.w, 1);
    return result;
}

__device__ void ConvexCell::new_triangle(uchar i, uchar j, uchar k) {
    if (nb_t+1 >= _MAX_T_) { 
        *status = triangle_overflow; 
        return; 
    }
    tr(nb_t) = make_uchar3(i, j, k);
    nb_t++;
}

__device__ int ConvexCell::new_point(int vid) {
    if (nb_v >= _MAX_P_) { 
        *status = vertex_overflow; 
        return -1; 
    }

    float4 B = point_from_ptr3(pts + 3 * vid);
    float4 dir = minus4(voro_seed, B);
    float4 ave2 = plus4(voro_seed, B);
    float dot = dot3(ave2,dir); // TODO safe to put dot4 here, dir.w = 0
    clip(nb_v) = make_float4(dir.x, dir.y, dir.z, -dot / 2.f);
    nb_v++;
    return nb_v - 1;
}

__device__ void ConvexCell::compute_boundary() {
    // clean circular list of the boundary
    FOR(i, _MAX_P_) boundary_next(i) = END_OF_LIST;
    first_boundary_ = END_OF_LIST;

    int nb_iter = 0;
    uchar t = nb_t;
    while (nb_r>0) {
        if (nb_iter++>100) { 
            *status = inconsistent_boundary; 
            return; 
        }
        bool is_in_border[3];
        bool next_is_opp[3];
        FOR(e, 3)   is_in_border[e] = (boundary_next(ith_plane(t, e)) != END_OF_LIST);
        FOR(e, 3)   next_is_opp[e] = (boundary_next(ith_plane(t, (e + 1) % 3)) == ith_plane(t, e));

        bool new_border_is_simple = true;
        // check for non manifoldness
        FOR(e, 3) if (!next_is_opp[e] && !next_is_opp[(e + 1) % 3] && is_in_border[(e + 1) % 3]) new_border_is_simple = false;

        // check for more than one boundary ... or first triangle
        if (!next_is_opp[0] && !next_is_opp[1] && !next_is_opp[2]) {
            if (first_boundary_ == END_OF_LIST) {
                FOR(e, 3) boundary_next(ith_plane(t, e)) = ith_plane(t, (e + 1) % 3);
                first_boundary_ = tr(t).x;
            }
            else new_border_is_simple = false;
        }

        if (!new_border_is_simple) {
            t++;
            if (t == nb_t + nb_r) t = nb_t;
            continue;
        }

        // link next
        FOR(e, 3) if (!next_is_opp[e]) boundary_next(ith_plane(t, e)) = ith_plane(t, (e + 1) % 3);

        // destroy link from removed vertices
        FOR(e, 3)  if (next_is_opp[e] && next_is_opp[(e + 1) % 3]) {
            if (first_boundary_ == ith_plane(t, (e + 1) % 3)) first_boundary_ = boundary_next(ith_plane(t, (e + 1) % 3));
            boundary_next(ith_plane(t, (e + 1) % 3)) = END_OF_LIST;
        }

        //remove triangle from R, and restart iterating on R
        swap(tr(t), tr(nb_t+nb_r-1));
        t = nb_t;
        nb_r--;
    }
}

__device__ void  ConvexCell::clip_by_plane(int vid) {
    int cur_v = new_point(vid); // add new plane equation
    if (*status == vertex_overflow) return;
    float4 eqn = clip(cur_v);
    nb_r = 0;

    int i = 0;
    while (i < nb_t) { // for all vertices of the cell
        float4 pc = compute_triangle_point(tr(i), false); // get the vertex
        if (dot4(eqn, pc)>0) { // is it clipped? then remove from T and place to R
            nb_t--;
            swap(tr(i), tr(nb_t));
            nb_r++;
        }
        else i++;
    }

    if (nb_r == 0) { // if no clips, then remove the plane equation
        nb_v--;
        return;
    }

    // Step 2: compute cavity boundary
    compute_boundary();
    if (*status != success) return;
    if (first_boundary_ == END_OF_LIST) return;

    // Step 3: Triangulate cavity
    uchar cir = first_boundary_;
    do {
        new_triangle(cur_v, cir, boundary_next(cir));
        if (*status != success) return;
        cir = boundary_next(cir);
    } while (cir != first_boundary_);
}

__device__ void get_tet_decomposition_of_vertex(ConvexCell& cc, int t, float4* P) {
    float4 C = cc.voro_seed;
    float4 A = cc.compute_triangle_point(tr(t));
    FOR(i,3)  P[2*i  ] = project_on_plane(C, clip(cc.ith_plane(t,i)));
    FOR(i, 3) P[2*i+1] = project_on_plane(A, plane_from_point_and_normal(C, cross3(minus4(P[2*i], C), minus4(P[(2*(i+1))%6], C))));
}

__device__ void export_bary_and_volume(ConvexCell& cc, float* out_pts, int seed) {
    float4 tet_bary; 
    float tet_vol;
    float4 bary_sum = make_float4(0, 0, 0, 0); 
    float cell_vol = 0;
    float4 P[6];
    float4 C = cc.voro_seed;

    FOR(t, cc.nb_t) {
        float4 A = cc.compute_triangle_point(tr(t));
        get_tet_decomposition_of_vertex(cc, t, P);
        FOR(i, 6) {
            get_tet_volume_and_barycenter(tet_bary, tet_vol, P[i], P[(i + 1) % 6], C, A);
            bary_sum = plus4(bary_sum, mul3(tet_vol, tet_bary));
            cell_vol += tet_vol;
        }
    }
    out_pts[3*seed    ] = bary_sum.x / cell_vol;
    out_pts[3*seed + 1] = bary_sum.y / cell_vol;
    out_pts[3*seed + 2] = bary_sum.z / cell_vol;
}

//###################  KERNEL   ######################
__device__ void compute_voro_cell(float * pts, int nbpts, unsigned int* neigs, Status* gpu_stat, float* out_pts, int seed) {
    FOR(d, 3) out_pts[3 * seed + d] = pts[3 * seed + d];

    //create BBox
    ConvexCell cc(seed, pts, &(gpu_stat[seed]));

    FOR(v, _K_) {
	 unsigned int z = neigs[_K_ * seed + v];
        cc.clip_by_plane(z);
//#ifndef __CUDA_ARCH__
      if (cc.is_security_radius_reached(point_from_ptr3(pts + 3*z))) {
          break;
      }
//#endif
        if (gpu_stat[seed] != success) {
            return;
        }
    }
    // check security radius
    if (!cc.is_security_radius_reached(point_from_ptr3(pts + 3 * neigs[_K_ * (seed+1) -1]))) {
        gpu_stat[seed] = security_radius_not_reached;
    }

    export_bary_and_volume(cc, out_pts, seed);
}

//----------------------------------KERNEL
__global__ void voro_cell_test_GPU_param(float * pts, int nbpts, unsigned int* neigs, Status* gpu_stat, float* out_pts) {
    int seed = blockIdx.x * blockDim.x + threadIdx.x;
    if (seed >= nbpts) return;
    compute_voro_cell(pts, nbpts, neigs, gpu_stat, out_pts, seed);
}

//----------------------------------WRAPPER
template <class T> struct GPUBuffer {
    void init(T* data) {
        IF_VERBOSE(std::cerr << "GPU: " << size * sizeof(T)/1048576 << " Mb used" << std::endl);
        cpu_data = data;
        cuda_check(hipMalloc((void**)& gpu_data, size * sizeof(T)));
        cpu2gpu();
    }
    GPUBuffer(std::vector<T>& v) {size = v.size() ;init(v.data());}
    ~GPUBuffer() { cuda_check(hipFree(gpu_data)); }

    void cpu2gpu() { cuda_check(hipMemcpy(gpu_data, cpu_data, size * sizeof(T), hipMemcpyHostToDevice)); }
    void gpu2cpu() { cuda_check(hipMemcpy(cpu_data, gpu_data, size * sizeof(T), hipMemcpyDeviceToHost)); }

    T* cpu_data;
    T* gpu_data;
    int size;
};

char StatusStr[5][128] = { "triangle_overflow","vertex_overflow ","inconsistent_boundary "," security_radius_not_reached ","success" };
void show_status_stats(std::vector<Status> &stat) {
    IF_VERBOSE(std::cerr << " \n\n\n---------Summary of success/failure------------\n");
    std::vector<int> nb_statuss(5, 0);
    FOR(i, stat.size()) nb_statuss[stat[i]]++;
    IF_VERBOSE(FOR(r, 5) std::cerr << " " << StatusStr[r] << "   " << nb_statuss[r] << "\n";)
        std::cerr << " " << StatusStr[4] << "   " << nb_statuss[4] << " /  " << stat.size() << "\n";
}

void cuda_check_error() {
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) { fprintf(stderr, "Failed (1) (error code %s)!\n", hipGetErrorString(err)); exit(EXIT_FAILURE); }
}

void compute_voro_diagram_GPU(std::vector<float>& pts, std::vector<Status> &stat, std::vector<float>& bary,int nb_Lloyd_iter) {
    int nbpts = pts.size() / 3;
    kn_problem *kn = NULL;
    {
        IF_VERBOSE(Stopwatch W("GPU KNN"));
        kn = kn_prepare((float3*) pts.data(), nbpts);
        hipMemcpy(pts.data(), kn->d_stored_points, kn->allocated_points * sizeof(float) * 3, hipMemcpyDeviceToHost);
        cuda_check_error();
        kn_solve(kn);
        IF_VERBOSE(kn_print_stats(kn));
    }

    GPUBuffer<float> out_pts_w(bary);
    GPUBuffer<Status> gpu_stat(stat);
//  if (nb_Lloyd_iter == 0) {
        IF_VERBOSE(Stopwatch W("GPU voro kernel only"));

        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start);

        voro_cell_test_GPU_param << < nbpts / VORO_BLOCK_SIZE + 1, VORO_BLOCK_SIZE >> > ((float*)kn->d_stored_points, nbpts, kn->d_knearests, gpu_stat.gpu_data, out_pts_w.gpu_data);
        cuda_check_error();

        hipEventRecord(stop);
        hipEventSynchronize(stop);
        float milliseconds = 0;
        hipEventElapsedTime(&milliseconds, start, stop);
        IF_VERBOSE(std::cerr << "kn voro: " << milliseconds << " msec" << std::endl);
//  }

//  // Lloyd
//  FOR(lit,nb_Lloyd_iter){
//      IF_VERBOSE(Stopwatch W("Loyd iterations"));
//      hipEvent_t start, stop;
//      hipEventCreate(&start);
//      hipEventCreate(&stop);
//      hipEventRecord(start);

//      voro_cell_test_GPU_param << < nbpts / VORO_BLOCK_SIZE + 1, VORO_BLOCK_SIZE >> > ((float*)kn->d_stored_points, nbpts, kn->d_knearests, gpu_stat.gpu_data, out_pts_w.gpu_data);
//      cuda_check_error();

//      voro_cell_test_GPU_param << < nbpts / VORO_BLOCK_SIZE + 1, VORO_BLOCK_SIZE >> > (out_pts_w.gpu_data, nbpts, kn->d_knearests, gpu_stat.gpu_data, (float*)kn->d_stored_points);
//      cuda_check_error();


//      hipEventRecord(stop);
//      hipEventSynchronize(stop);
//      float milliseconds = 0;
//      hipEventElapsedTime(&milliseconds, start, stop);
//      IF_VERBOSE(std::cerr << "kn voro: " << milliseconds << " msec" << std::endl);
//  }

    {
        IF_VERBOSE(Stopwatch W("copy data back to the cpu"));
        out_pts_w.gpu2cpu();
        gpu_stat.gpu2cpu();
    }

    kn_free(&kn);
    show_status_stats(stat);
}

