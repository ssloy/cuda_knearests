#include "hip/hip_runtime.h"
#include <vector>
#include <iostream>
#include <fstream>
#include <sstream>
#include <cassert>
#include <algorithm>
#include "VBW.h"
#include <set>

#include "knearests.h"
const int DEFAULT_NB_PLANES = 35; // touche pas à ça


bool load_file(const char* filename, std::vector<float>& xyz) {
    std::ifstream in;
    in.open (filename, std::ifstream::in);
    if (in.fail()) return false;
    std::string line;
    int npts = 0;
    bool firstline = true;
    float x,y,z;
    while (!in.eof()) {
        std::getline(in, line);
        if (!line.length()) continue;
        std::istringstream iss(line.c_str());
        if (firstline) {
            iss >> npts;
            firstline = false;
        } else {
            iss >> x >> y >> z;
            xyz.push_back(x);
            xyz.push_back(y);
            xyz.push_back(z);
        }
    }
    assert(xyz.size() == npts*3);
    in.close();
    return true;
}

void get_bbox(const std::vector<float>& xyz, float& xmin, float& ymin, float& zmin, float& xmax, float& ymax, float& zmax) {
    int nb_v = xyz.size()/3;
    xmin = xmax = xyz[0];
    ymin = ymax = xyz[1];
    zmin = zmax = xyz[2];
    for(int i=1; i<nb_v; ++i) {
        xmin = std::min(xmin, xyz[3*i]);
        ymin = std::min(ymin, xyz[3*i+1]);
        zmin = std::min(zmin, xyz[3*i+2]);
        xmax = std::max(xmax, xyz[3*i]);
        ymax = std::max(ymax, xyz[3*i+1]);
        zmax = std::max(zmax, xyz[3*i+2]);	    
    }
    float d = xmax-xmin;
    d = std::max(d, ymax-ymin);
    d = std::max(d, zmax-zmin);
    d = 0.001f*d;
    xmin -= d;
    ymin -= d;
    zmin -= d;
    xmax += d;
    ymax += d;
    zmax += d;
}




float dot_3d(float* v0, float* v1) { return v0[0] * v1[0] + v0[1] * v1[1] + v0[2] * v1[2]; }
float length2_3d(float* v0) { return dot_3d(v0,v0); }
struct Dist2SeedCmp {
	Dist2SeedCmp(std::vector<float>& p_points, int p_seed) : points(p_points) { seed = p_seed; }
	bool operator()(int id0, int id1) {
		float v0[3];FOR(d, 3) v0[d]=points[3 * seed + d] - points[3 * id0 + d];
		float v1[3];FOR(d, 3) v1[d] = points[3 * seed + d] - points[3 * id1 + d];
		return length2_3d(&(v0[0])) < length2_3d(&(v1[0]));
	}
	std::vector<float>& points;
	int seed;
};

int main(int argc, char** argv) {
	if (2 > argc) {
		std::cerr << "Usage: " << argv[0] << " points.xyz" << std::endl;
		return 1;
	}

	std::vector<float> points;
	std::vector<int> neighbors;
	std::vector<double> watch(1, now());
	{
		if (!load_file(argv[1], points)) {
			std::cerr << argv[1] << ": could not load file" << std::endl;
			return 1;
		}
		int NBPTS = points.size() / 3;
		points.resize(3 * NBPTS);
		FOR(i, 3 * NBPTS) points[i] = double(rand()) / RAND_MAX;
	}

	{ // normalize point cloud between [0,1000]^3
		float xmin, ymin, zmin, xmax, ymax, zmax;
		get_bbox(points, xmin, ymin, zmin, xmax, ymax, zmax);

		float maxside = std::max(std::max(xmax - xmin, ymax - ymin), zmax - zmin);
		for (int i = 0; i < points.size() / 3; i++) {
			points[i * 3 + 0] = (points[i * 3 + 0] - xmin) / maxside;
			points[i * 3 + 1] = (points[i * 3 + 1] - ymin) / maxside;
			points[i * 3 + 2] = (points[i * 3 + 2] - zmin) / maxside;
		}
		for (int i = 0; i < points.size(); i++) {
			points[i] *= 1000.;
		}
		get_bbox(points, xmin, ymin, zmin, xmax, ymax, zmax);
		std::cerr << "bbox [" << xmin << ":" << xmax << "], [" << ymin << ":" << ymax << ", [" << zmin << ":" << zmax << "]" << std::endl;
	}

	std::cerr << "\n----------------------------------------pointset loaded in " << now() - watch.back() << " seconds\n"; watch.push_back(now());

	{ // solve kn problem
		neighbors = std::vector<int>(points.size() / 3 * DEFAULT_NB_PLANES+1, -1);
		std::cerr << "\n----------------------------------------memory for neig reserved in " << now() - watch.back() << " seconds\n"; watch.push_back(now());
		kn_problem *kn = kn_prepare(points.data(), points.size() / 3);
		std::cerr << "\n----------------------------------------KNN struct prepared  in " << now() - watch.back() << " seconds\n"; watch.push_back(now());
		kn_solve(kn);
		std::cerr << "\n----------------------------------------KNN precomputed in " << now() - watch.back() << " seconds\n"; watch.push_back(now());

		kn_iterator *it = kn_begin_enum(kn); // retrieve neighbors, skip the point itself
		for (int v = 0; v < points.size() / 3; v++) {
			unsigned int knpt = kn_first_nearest_id(it, v);
			int j = 0;
			while (knpt != UINT_MAX) {
				if (v != knpt) {
					neighbors[v*DEFAULT_NB_PLANES + j] = knpt;
					j++;
				}
				knpt = kn_next_nearest_id(it);
			}
			assert(j == DEFAULT_NB_PLANES);
		}

		// the data was re-ordered, so retreive it from the GPU
		float *fp = kn_point(it, 0);
		for (int v = 0; v < points.size(); v++) {
			points[v] = fp[v];
		}

	kn_print_stats(kn);
	kn_check_for_dupes(kn);
	kn_free(&kn);

	//        kn_sanity_check(kn); // very slow sanity checks
}


	std::cerr << "\n-----------------------------sort neigs\n";
	int nb_voro_cells = points.size() / 3;


	FOR(i, nb_voro_cells) {
		Dist2SeedCmp cmp(points,i);
		std::sort(&(neighbors[i*DEFAULT_NB_PLANES]), &(neighbors[(i+1)*DEFAULT_NB_PLANES]),cmp);
	}



    std::vector<int> tets(4*MAX_T* points.size() / 3,-1);
    watch.push_back(now());
    std::cerr << "\n-----------------------------start voro\n" ;




    compute_voro_diagram(points.data(), points.size() / 3, neighbors.data(), 0, 0, 0, 1000, 1000, 1000, tets.data());
    //FOR(i, nb_voro_cells) voro_cell(i, points.data(), points.size() / 3, neighbors.data(), 0, 0, 0, 1000, 1000, 1000, tets.data(),false);
    
    std::cerr << "\n----------------------------------------Voro computed in " << now() - watch.back() << " seconds\n"; watch.push_back(now());

    int nb_real_tets = 0;
    FOR(i, nb_voro_cells) FOR(j, MAX_T) if (tets[i * 4 * MAX_T + 4 * j] != -1
	    && tets[i * 4 * MAX_T + 4 * j] > tets[i * 4 * MAX_T + 4 * j+1] 
	    && tets[i * 4 * MAX_T + 4 * j] > tets[i * 4 * MAX_T + 4 * j + 2]
	    && tets[i * 4 * MAX_T + 4 * j] > tets[i * 4 * MAX_T + 4 * j + 3]
	    )nb_real_tets++;
    


    std::ofstream out("out.tet");
    out << points.size() / 3 << " vertices" << std::endl;
    out << nb_real_tets << " tets" << std::endl;
    FOR(v,points.size()/3)   out << points[3 * v] << " "<< points[3 * v + 1] << " "<< points[3 * v + 2] << std::endl;
    

    FOR(i, nb_voro_cells) {
	    int offset = i * 4 * MAX_T;
	    FOR(j, MAX_T) if (tets[offset + 4 * j] != -1
		    && tets[i * 4 * MAX_T + 4 * j] > tets[i * 4 * MAX_T + 4 * j + 1]
		    && tets[i * 4 * MAX_T + 4 * j] > tets[i * 4 * MAX_T + 4 * j + 2]
		    && tets[i * 4 * MAX_T + 4 * j] > tets[i * 4 * MAX_T + 4 * j + 3]
		    ) {
		    out << "4 "<<tets[offset + 4 * j] << " " << tets[offset + 4 * j + 1] << " " << tets[offset + 4 * j + 2] << " " << tets[offset + 4 * j + 3] << " \n";
	    }
    }

    { // re-check for dupes
        for (int v=0; v<points.size()/3; v++) {
            std::set<int> kns;
            for (int i=0; i<DEFAULT_NB_PLANES; i++) {
                int kni = neighbors[v*DEFAULT_NB_PLANES+i];
                if (kni < UINT_MAX) {
                    if (kns.find(kni) != kns.end()) {
                        std::cerr << "ERROR duplicated entry for point " << v << std::endl;
                        return 1;
                    }
                    kns.insert(kni);
                }
            }
        }
    }


    return 0;
}

