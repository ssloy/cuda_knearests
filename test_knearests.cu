#include "hip/hip_runtime.h"
#include <vector>
#include <algorithm>
#include <iostream>
#include <fstream>
#include <sstream>
#include <cassert>
#include <set>

#include "knearests.h"
#include "kd_tree.h"

#if defined(__linux__)
#   include <sys/times.h>
#endif

class Stopwatch {
    public:
        Stopwatch(const char* taskname) :
            taskname_(taskname), start_(now()) {
                std::cout << taskname_ << "..." << std::endl;
            }
        ~Stopwatch() {
            double elapsed = now() - start_;
            std::cout << taskname_ << ": "
                << elapsed << "s" << std::endl;
        }
        static double now() {
#if defined(__linux__)
            tms now_tms;
            return double(times(&now_tms)) / 100.0;
#elif defined(WIN32) || defined(_WIN64)
            return double(GetTickCount()) / 1000.0;	    
#else
            return 0.0;
#endif	    
        }
    private:
        const char* taskname_;
        double start_;
};

bool load_file(const char* filename, std::vector<float>& xyz) {
    std::ifstream in;
    in.open (filename, std::ifstream::in);
    if (in.fail()) return false;
    std::string line;
    int npts = 0;
    bool firstline = true;
    float x,y,z;
    while (!in.eof()) {
        std::getline(in, line);
        if (!line.length()) continue;
        std::istringstream iss(line.c_str());
        if (firstline) {
            iss >> npts;
            firstline = false;
        } else {
            iss >> x >> y >> z;
            xyz.push_back(x);
            xyz.push_back(y);
            xyz.push_back(z);
        }
    }
    assert(xyz.size() == npts*3);
    in.close();
    return true;
}

void get_bbox(const std::vector<float>& xyz, float& xmin, float& ymin, float& zmin, float& xmax, float& ymax, float& zmax) {
    int nb_v = xyz.size()/3;
    xmin = xmax = xyz[0];
    ymin = ymax = xyz[1];
    zmin = zmax = xyz[2];
    for(int i=1; i<nb_v; ++i) {
        xmin = std::min(xmin, xyz[3*i]);
        ymin = std::min(ymin, xyz[3*i+1]);
        zmin = std::min(zmin, xyz[3*i+2]);
        xmax = std::max(xmax, xyz[3*i]);
        ymax = std::max(ymax, xyz[3*i+1]);
        zmax = std::max(zmax, xyz[3*i+2]);	    
    }
    float d = xmax-xmin;
    d = std::max(d, ymax-ymin);
    d = std::max(d, zmax-zmin);
    d = 0.001f*d;
    xmin -= d;
    ymin -= d;
    zmin -= d;
    xmax += d;
    ymax += d;
    zmax += d;
}

int main(int argc, char** argv) {
    if (2>argc) {
        std::cerr << "Usage: " << argv[0] << " points.xyz" << std::endl;
        return 1;
    }
    
    std::vector<float> points;
    const int DEFAULT_NB_PLANES = 36; // touche pas à ça
    std::vector<int> neighbors;

    { // load point cloud file
        if (!load_file(argv[1], points)) {
            std::cerr << argv[1] << ": could not load file" << std::endl;
            return 1;
        }
        for (int i=0; i<points.size(); i++) {
            points[i] = rand()/(float)RAND_MAX;
        }
    }

    { // normalize point cloud between [0,1000]^3
        float xmin,ymin,zmin,xmax,ymax,zmax;
        get_bbox(points, xmin, ymin, zmin, xmax, ymax, zmax);

        float maxside = std::max(std::max(xmax-xmin, ymax-ymin), zmax-zmin);
        for (int i=0; i<points.size()/3; i++) {
            points[i*3+0] = (points[i*3+0]-xmin)/maxside;
            points[i*3+1] = (points[i*3+1]-ymin)/maxside;
            points[i*3+2] = (points[i*3+2]-zmin)/maxside;
        }
        for (int i=0; i<points.size(); i++) {
            points[i] *= 1000.;
        }
        get_bbox(points, xmin, ymin, zmin, xmax, ymax, zmax);
        std::cerr << "bbox [" << xmin << ":" << xmax << "], [" << ymin << ":" << ymax << ", [" << zmin << ":" << zmax << "]" << std::endl;
    }

    { // solve kn problem
        Stopwatch W("knn gpu");

        std::vector<int> neighbors_perm = std::vector<int>(points.size()/3*DEFAULT_NB_PLANES, -1);
        kn_problem *kn = kn_prepare(points.data(), points.size()/3);
        kn_solve(kn);

        kn_iterator *it = kn_begin_enum(kn); // retrieve neighbors, skip the point itself
        for (int v=0; v<points.size()/3; v++) {
            unsigned int knpt = kn_first_nearest_id(it,v);
            int j = 0;
            while (knpt!=UINT_MAX) {
                neighbors_perm[v*DEFAULT_NB_PLANES + j++] = knpt;
                knpt = kn_next_nearest_id(it);
            }
            assert(j==DEFAULT_NB_PLANES);
        }

/*
        // the data was re-ordered, so retreive it from the GPU
        float *fp = kn_point(it, 0); 
        for (int v=0; v<points.size(); v++) {
            points[v] = fp[v];
        }
        */
        kn_print_stats(kn);

        int nb_points = points.size()/3;
        std::vector<int> neighbors_tmp = std::vector<int>(neighbors_perm.size());
        neighbors = std::vector<int>(neighbors_perm.size());
        unsigned int *permutation = kn_get_permutation(kn);

        for (int i=0; i<neighbors_perm.size(); i++) {
            neighbors_tmp[i] = permutation[1+neighbors_perm[i]];
        }
        for (int i=0; i<nb_points; i++) {
            for (int j=0; j<DEFAULT_NB_PLANES; j++) {
                neighbors[permutation[1+i]*DEFAULT_NB_PLANES+j] = neighbors_tmp[i*DEFAULT_NB_PLANES+j];
            }
        }

        { // sanity check for the permutation array
            std::sort(permutation, permutation+nb_points+1);
            assert(permutation[0]==0);
            for (int i=0; i<nb_points-1; i++) {
                assert(permutation[i]+1 == permutation[i+1]);
            }
        }
        free(permutation);

//        kn_sanity_check(kn); // very slow sanity checks

        kn_free(&kn);
    }

    { // re-check for dupes
        std::cerr << "checking for dupes...";
#pragma omp parallel for
        for (int v=0; v<points.size()/3; v++) {
            std::set<int> kns;
            for (int i=0; i<DEFAULT_NB_PLANES; i++) {
                int kni = neighbors[v*DEFAULT_NB_PLANES+i];
                if (kni < UINT_MAX) {
                    if (kns.find(kni) != kns.end()) {
                        std::cerr << "ERROR: duplicated entry for point " << v << std::endl;
                        break;
                    }
                    kns.insert(kni);
                }
            }
        }
        std::cerr << "ok" << std::endl;
    }

    std::cerr << "Building KD-tree...";
    int nb_points = points.size()/3;
    std::vector<int> cpu_neighbors(nb_points*DEFAULT_NB_PLANES);
    KdTree KD(3);
    {
        Stopwatch W("Build kd-tree");
        KD.set_points(nb_points, points.data());
        std::cerr << "ok" << std::endl << "Querying the KD-tree...";

#pragma omp parallel for
        for (int v=0; v<nb_points; ++v) {
            int neigh[DEFAULT_NB_PLANES+1];
            float sq_dist[DEFAULT_NB_PLANES+1];	
            KD.get_nearest_neighbors(DEFAULT_NB_PLANES+1,v,neigh,sq_dist);

            for(int j=0; j<DEFAULT_NB_PLANES; ++j) {
                cpu_neighbors[v*DEFAULT_NB_PLANES+j] = neigh[j+1];
            }
        }
        std::cerr << "ok" << std::endl;
    }
    std::cerr << "Comparing CPU and GPU versions...";
    for (int i=0; i<nb_points; i++) {
        std::sort(    neighbors.begin()+i*DEFAULT_NB_PLANES,     neighbors.begin()+(i+1)*DEFAULT_NB_PLANES);
        std::sort(cpu_neighbors.begin()+i*DEFAULT_NB_PLANES, cpu_neighbors.begin()+(i+1)*DEFAULT_NB_PLANES);
    }
    for (int i=0; i<nb_points; i++) {
        for (int j=0; j<DEFAULT_NB_PLANES; j++) {
            if (cpu_neighbors[i*DEFAULT_NB_PLANES+j]==neighbors[i*DEFAULT_NB_PLANES+j]) continue;
            std::cerr << "Error in point " << i << " neigbor " << j << std::endl;
            for (int k=0; k<DEFAULT_NB_PLANES; k++) {
                std::cerr << cpu_neighbors[i*DEFAULT_NB_PLANES+k] << "-" << neighbors[i*DEFAULT_NB_PLANES+k] << std::endl;
            }
            assert(false);
        }
    }
    std::cerr << "ok" << std::endl;

    return 0;
}

