#include "hip/hip_runtime.h"
#include <vector>
#include <algorithm>
#include <iostream>
#include <fstream>
#include <sstream>
#include <cassert>



#include "params.h"
#include "VBW.h"
#include "stopwatch.h"



void get_bbox(const std::vector<float>& xyz, float& xmin, float& ymin, float& zmin, float& xmax, float& ymax, float& zmax) {
    int nb_v = xyz.size()/3;
    xmin = xmax = xyz[0];
    ymin = ymax = xyz[1];
    zmin = zmax = xyz[2];
    for(int i=1; i<nb_v; ++i) {
        xmin = std::min(xmin, xyz[3*i]);
        ymin = std::min(ymin, xyz[3*i+1]);
        zmin = std::min(zmin, xyz[3*i+2]);
        xmax = std::max(xmax, xyz[3*i]);
        ymax = std::max(ymax, xyz[3*i+1]);
        zmax = std::max(zmax, xyz[3*i+2]);	    
    }
    float d = xmax-xmin;
    d = std::max(d, ymax-ymin);
    d = std::max(d, zmax-zmin);
    d = 0.001f*d;
    xmin -= d;
    ymin -= d;
    zmin -= d;
    xmax += d;
    ymax += d;
    zmax += d;
}

bool load_file(const char* filename, std::vector<float>& xyz, bool normalize=true) {
    std::ifstream in;
    in.open(filename, std::ifstream::in);
    if (in.fail()) return false;
    std::string line;
    int npts = 0;
    bool firstline = true;
    float x,y,z;
    while (!in.eof()) {
        std::getline(in, line);
        if (!line.length()) continue;
        std::istringstream iss(line.c_str());
        if (firstline) {
            iss >> npts;
            firstline = false;
        } else {
            iss >> x >> y >> z;
            xyz.push_back(x);
            xyz.push_back(y);
            xyz.push_back(z);
        }
    }
    assert(xyz.size() == npts*3);
    in.close();



    normalize = false;
    IF_NORMALIZE_PTS(normalize = true;)


    if (normalize) { // normalize point cloud between [0,1000]^3
        float xmin,ymin,zmin,xmax,ymax,zmax;
        get_bbox(xyz, xmin, ymin, zmin, xmax, ymax, zmax);

        float maxside = std::max(std::max(xmax-xmin, ymax-ymin), zmax-zmin);
#pragma omp parallel for
        for (int i=0; i<xyz.size()/3; i++) {
            xyz[i*3+0] = 1000.f*(xyz[i*3+0]-xmin)/maxside;
            xyz[i*3+1] = 1000.f*(xyz[i*3+1]-ymin)/maxside;
            xyz[i*3+2] = 1000.f*(xyz[i*3+2]-zmin)/maxside;
        }
        get_bbox(xyz, xmin, ymin, zmin, xmax, ymax, zmax);
        std::cerr << "bbox [" << xmin << ":" << xmax << "], [" << ymin << ":" << ymax << ", [" << zmin << ":" << zmax << "]" << std::endl;
    }
    return true;
}

void export_tet_mesh(float* pts, int nb_pts, int* tets, int nb_tets){
    Stopwatch W("output out.tet");
    std::ofstream out("C:\\DATA\\out.tet");
    out << nb_pts << " vertices" << std::endl;
    out << nb_tets << " tets" << std::endl;
    FOR(v, nb_pts)   out << pts[3 * v] << " " << pts[3 * v + 1] << " " << pts[3 * v + 2] << std::endl;
    FOR(j, nb_tets)  out << "4 " << tets[4 * j] << " " << tets[4 * j + 1] << " " << tets[4 * j + 2] << " " << tets[4 * j + 3] << " \n";
}

void printDevProp() {
    int devCount; // Number of CUDA devices
    hipGetDeviceCount(&devCount);
    printf("CUDA Device Query...\n");
    printf("There are %d CUDA devices.\n", devCount);

    // Iterate through devices
    for (int i=0; i<devCount; ++i) {
        // Get device properties
        printf("\nCUDA Device #%d\n", i);
        hipDeviceProp_t devProp;
        hipGetDeviceProperties(&devProp, i);
        printf("Major revision number:         %d\n",  devProp.major);
        printf("Minor revision number:         %d\n",  devProp.minor);
        printf("Name:                          %s\n",  devProp.name);
        printf("Total global memory:           %u\n",  devProp.totalGlobalMem);
        printf("Total shared memory per block: %u\n",  devProp.sharedMemPerBlock);
        printf("Total registers per block:     %d\n",  devProp.regsPerBlock);
        printf("Warp size:                     %d\n",  devProp.warpSize);
        printf("Maximum memory pitch:          %u\n",  devProp.memPitch);
        printf("Maximum threads per block:     %d\n",  devProp.maxThreadsPerBlock);
        for (int i = 0; i < 3; ++i)
            printf("Maximum dimension %d of block:  %d\n", i, devProp.maxThreadsDim[i]);
        for (int i = 0; i < 3; ++i)
            printf("Maximum dimension %d of grid:   %d\n", i, devProp.maxGridSize[i]);
        printf("Clock rate:                    %d\n",  devProp.clockRate);
        printf("Total constant memory:         %u\n",  devProp.totalConstMem);
        printf("Texture alignment:             %u\n",  devProp.textureAlignment);
        printf("Concurrent copy and execution: %s\n",  (devProp.deviceOverlap ? "Yes" : "No"));
        printf("Number of multiprocessors:     %d\n",  devProp.multiProcessorCount);
        printf("Kernel execution timeout:      %s\n",  (devProp.kernelExecTimeoutEnabled ? "Yes" : "No"));
    }
}


void drop_xyz_file(std::vector<float>& pts) {
    std::fstream file;
    static int fileid = 0;
    char filename[1024];
    sprintf(filename, "C:\\DATA\\dr_%d_.xyz", fileid);
    fileid++;
    file.open(filename, std::ios_base::out);
    file << pts.size() / 3 << std::endl;
    FOR(i, pts.size() / 3) file << pts[3 * i] << "  " << pts[3 * i + 1] << "  " << pts[3 * i + 2] << " \n";
    file.close();
}
void drop_xyz_file(std::vector<float>& pts, std::vector<bool>& heavy) {
    std::fstream file;
    static int fileid = -1;
    fileid++;
    char filename[1024];

    sprintf(filename, "C:\\DATA\\o_%03d_heavy.xyz", fileid);
    file.open(filename, std::ios_base::out);
    int nbheavy = 0;
    FOR(i, heavy.size()) if (heavy[i]) nbheavy++;
    file << nbheavy << std::endl;
    FOR(i, pts.size() / 3) if (heavy[i]) file << pts[3 * i] << "  " << pts[3 * i + 1] << "  " << pts[3 * i + 2] << " \n";
    file.close();
  //  return;
    sprintf(filename, "C:\\DATA\\o_%03d_light.xyz", fileid);
    file.open(filename, std::ios_base::out);
    file << heavy.size() - nbheavy << std::endl;
    FOR(i, pts.size() / 3) if (!heavy[i]) file << pts[3 * i] << "  " << pts[3 * i + 1] << "  " << pts[3 * i + 2] << " \n";
    file.close();
  
}




int main(int argc, char** argv) {
    printDevProp();
    if (2>argc) {
        std::cerr << "Usage: " << argv[0] << " points.xyz" << std::endl;
        return 1;
    }
    
    std::vector<float> pts;

    if (!load_file(argv[1], pts)) {
        std::cerr << argv[1] << ": could not load file" << std::endl;
        return 1;
    }
    //pts.resize(9000);
    //FOR(i, pts.size()) pts[i] =  1000.*double(rand()) / RAND_MAX;

    /*
    int n=216;
    pts.resize(n*n*n*3);
    for (int x=0; x<n; x++) {
        for (int y=0; y<n; y++) {
            for (int z=0; z<n; z++) {
                float noise[3] = {0.};
                for (int i=0; i<3; i++) {
                    noise[i] = 3.*static_cast<float>(rand())/static_cast<float>(RAND_MAX);
                }
                pts[(x+y*n+z*n*n)*3+0] = x/static_cast<float>(n)*1000. + noise[0];
                pts[(x+y*n+z*n*n)*3+1] = y/static_cast<float>(n)*1000. + noise[1];
                pts[(x+y*n+z*n*n)*3+2] = z/static_cast<float>(n)*1000. + noise[2];
            }
        }
    }
*/

    int nb_pts = pts.size()/3;


    std::vector<int> tets(0);
    IF_OUTPUT_TET(tets.resize(nb_pts * 4 * 50);)
    int nb_tets = 0;

    std::vector<float> out_pts(0);
    IF_OUTPUT_P2BARY(out_pts.resize(pts.size(), 0);)

        std::vector<Status> stat(nb_pts);
   
#if STAT_MODE
        {// CPU test /debug/stat
            Stopwatch W("CPU run");
            compute_voro_diagram_CPU(pts, tets, nb_tets, stat, out_pts,NULL);
            IF_OUTPUT_TET(export_tet_mesh(pts.data(), nb_pts, tets.data(), nb_tets);)

        }
        return;
#endif

#if LOG_GPU_SPEED
        {// single GPU run
            int iter = 5; 
            Stopwatch W("GPU run");
            int block_size = pow(2, iter);
            std::cerr << " block_size = " << block_size << std::endl;
            compute_voro_diagram_GPU(pts, tets, nb_tets, stat, out_pts,NULL, block_size);
        }
        return;
#endif
        
#if LOYD_MODE
        {
            Stopwatch W("Test Loyd");
            FOR(it, 50) {
                compute_voro_diagram_GPU(pts, tets, nb_tets, stat, out_pts, NULL, 32);
               // FOR(i, out_pts.size()) out_pts[i] /= 10.;
                FOR(i, pts.size()) if (pts[i]+out_pts[i]<1000 && pts[i]+out_pts[i]>0 ) pts[i] = pts[i]+out_pts[i];
                if ((it%5)==0 )drop_xyz_file(pts);
            }
        }
        return;
#endif

#if SIMU_MODE
        {
            Stopwatch W("Fluids");
            unsigned int *permutation;
            std::vector<bool> heavy(nb_pts);
            std::vector<float> indir_buffer(3*nb_pts);
            //FOR(i, nb_pts) heavy[i] = (pts[3 * i] > 500);
            //FOR(i, nb_pts) heavy[i] = (pts[3 * i + 2] > 600 + 200.*cos(6.28*pts[3 * i + 1] / 1000.) + 200 * cos(6.28*pts[3 * i + 0] / 1000.));
            FOR(i, nb_pts) heavy[i] = (pts[3 * i + 2] > 600 - 200.*cos(6.28*pts[3 * i + 1] / 1000.) * cos(6.28*pts[3 * i + 0] / 1000.));
           // FOR(i, nb_pts) heavy[i] = (pts[3 * i + 2] > 600 + 300.*cos(6.28*pts[3 * i + 1] / 1000.) );
            //FOR(i, nb_pts) heavy[i] = (std::abs(pts[3 * i + 0] - 500)<200 && std::abs(pts[3 * i + 1] - 500)<200 && std::abs(pts[3 * i + 2] - 500)<200);

            int inf = 0; int sup = nb_pts - 1;
            while (inf < sup) {
                if (!heavy[inf] && heavy[sup]) {
                    FOR(d, 3) std::swap(pts[3 * inf + d], pts[3 * sup + d]);
                    heavy[inf] = true;
                    heavy[sup] = false;
                }
                if (heavy[inf]) inf++;
                if (!heavy[sup]) sup--;
            }
            std::cerr << "2*(nb heavy) " <<2* inf << "---> \n";


            std::vector<float> speed(3 * nb_pts,0);

            FOR(it, 1001) {
                std::cerr << "iter " << it << "---> ";
                if (it % 50 == 0)  
                    drop_xyz_file(pts);

                compute_voro_diagram_GPU(pts, tets, nb_tets, stat, out_pts, &permutation, 32);
                //compute_voro_diagram_CPU(pts, tets, nb_tets, stat, out_pts, &permutation);

                FOR(i, pts.size()) indir_buffer[3 * permutation[i / 3] + i % 3] = out_pts[i];
                FOR(i, pts.size()) out_pts[i] = indir_buffer[i];

                FOR(i, pts.size()) indir_buffer[3 * permutation[i / 3] + i % 3] = pts[i];
                FOR(i, pts.size()) pts[i] = indir_buffer[i];

                float dt = .1;
                FOR(i, pts.size()) {
                    float r = 50;
                    //if (heavy[i / 3]) r = 10;
                    
                    r *= float(nb_pts)/3000.;
                    r *= 25;

                    float a = r * out_pts[i] ;
                    //if (i % 3 == 2) a += -.01;

                    speed[i] = speed[i] + dt * a;
                    pts[i] = pts[i] + dt * speed[i];

                    if (pts[i] < 0) { speed[i] = 0; pts[i] = .1; }
                    if (pts[i] > 1000) { speed[i] = 0; pts[i] = 999.9; }
                }

                free(permutation);
            }
        }
#endif
    
    return 0;
}

